#include "hip/hip_runtime.h"

#include <cstdio>
#include <hipcub/hipcub.hpp>
#include <stdint.h>

template <typename T>
__device__ void bitwise_and(const T *d_in1, const T *d_in2, T *d_out,
                            const uint32_t N) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    d_out[idx] = d_in1[idx] & d_in2[idx];
  }
}

#define BITWISE_AND_OP(TYPENAME, RUST_NAME)                                    \
  extern "C" __global__ void bitwise_and_##RUST_NAME(                          \
      const TYPENAME *d_in1, const TYPENAME *d_in2, TYPENAME *d_out,           \
      uint32_t N) {                                                            \
    bitwise_and(d_in1, d_in2, d_out, N);                                       \
  }

BITWISE_AND_OP(uint8_t, u8)
BITWISE_AND_OP(uint32_t, u32)
BITWISE_AND_OP(int64_t, i64)

template <typename T>
__device__ void bitwise_or(const T *d_in1, const T *d_in2, T *d_out,
                           const uint32_t N) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    d_out[idx] = d_in1[idx] | d_in2[idx];
  }
}

#define BITWISE_OR_OP(TYPENAME, RUST_NAME)                                     \
  extern "C" __global__ void bitwise_or_##RUST_NAME(                           \
      const TYPENAME *d_in1, const TYPENAME *d_in2, TYPENAME *d_out,           \
      uint32_t N) {                                                            \
    bitwise_or(d_in1, d_in2, d_out, N);                                        \
  }

BITWISE_OR_OP(uint8_t, u8)
BITWISE_OR_OP(uint32_t, u32)
BITWISE_OR_OP(int64_t, i64)

template <typename T>
__device__ void bitwise_xor(const T *d_in1, const T *d_in2, T *d_out,
                            const uint32_t N) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    d_out[idx] = d_in1[idx] ^ d_in2[idx];
  }
}

#define BITWISE_XOR_OP(TYPENAME, RUST_NAME)                                    \
  extern "C" __global__ void bitwise_xor_##RUST_NAME(                          \
      const TYPENAME *d_in1, const TYPENAME *d_in2, TYPENAME *d_out,           \
      uint32_t N) {                                                            \
    bitwise_xor(d_in1, d_in2, d_out, N);                                       \
  }

BITWISE_XOR_OP(uint8_t, u8)
BITWISE_XOR_OP(uint32_t, u32)
BITWISE_XOR_OP(int64_t, i64)